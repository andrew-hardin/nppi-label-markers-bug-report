#include <cstdint>
#include <iostream>
#include <vector>

#include <nppi.h>

#define CHECK_CUDA(x) \
  if((x) != hipSuccess) exit(1)

#define CHECK_NPP(x) \
  if((x) != NPP_SUCCESS) exit(1)

static const int kWidth = 6;
static const int kHeight = 7;
static const NppiSize kSize = { kWidth, kHeight };

template<typename T>
void PrintGrid(const std::vector<T>& item) {
  size_t i = 0;
  for(int r = 0; r < kHeight; r++) {
    for(int c = 0; c < kWidth; c++) {
      std::cout << static_cast<int>(item[i++]) << ' ';
    }
    std::cout << '\n';
  }
}

int main() {

  std::vector<uint8_t> input_host = {
        0, 1, 1, 1, 0, 0,
        0, 1, 1, 1, 0, 0,
        0, 1, 1, 1, 0, 0,
        0, 0, 0, 0, 0, 0,
        1, 1, 0, 0, 1, 1,
        1, 1, 0, 0, 1, 1,
        1, 1, 0, 0, 1, 1,
  };

  // Input -> Device.
  void* input_device;
  CHECK_CUDA(hipMalloc(&input_device, sizeof(uint8_t) * input_host.size()));
  CHECK_CUDA(hipMemcpy(input_device, input_host.data(), sizeof(uint8_t) * input_host.size(), hipMemcpyKind::hipMemcpyHostToDevice));

  // Output.
  void* output_device;
  CHECK_CUDA(hipMalloc(&output_device, sizeof(uint32_t) * input_host.size()));

  // Tmp buffer.
  int tmp_buffer_size;
  CHECK_NPP(nppiLabelMarkersUFGetBufferSize_32u_C1R(kSize, &tmp_buffer_size));
  void* tmp_buffer;
  CHECK_CUDA(hipMalloc(&tmp_buffer, tmp_buffer_size));

  // Invoke.
  CHECK_NPP(nppiLabelMarkersUF_8u32u_C1R(
    static_cast<Npp8u*>(input_device), kWidth,
    static_cast<Npp32u*>(output_device), kWidth * sizeof(uint32_t),
    kSize, NppiNorm::nppiNormL1,
    static_cast<Npp8u*>(tmp_buffer)));

  // Copy result back to host.
  std::vector<uint32_t> output_host(input_host.size());
  CHECK_CUDA(hipMemcpy(output_host.data(), output_device, sizeof(uint32_t) * output_host.size(), hipMemcpyKind::hipMemcpyDeviceToHost));

  std::cout << "INPUT\n";
  std::cout << "-----\n";
  PrintGrid(input_host);

  std::cout << "\nOUTPUT\n";
  std::cout << "-----\n";
  PrintGrid(output_host);

  return 0;
}